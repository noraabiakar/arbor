#include "hip/hip_runtime.h"
#include <arbor/fvm_types.hpp>

#include "cuda_common.hpp"
#include "matrix_common.hpp"

namespace arb {
namespace gpu {

namespace kernels {
/// GPU implementatin of Hines matrix assembly
/// Flat layout
/// For a given time step size dt
///     - use the precomputed alpha and alpha_d values to construct the diagonal
///       and off diagonal of the symmetric Hines matrix.
///     - compute the RHS of the linear system to solve
template <typename T, typename I>
__global__
void assemble_matrix_flat(
        T* d,
        T* rhs,
        const T* invariant_d,
        const T* voltage,
        const T* current,
        const T* cv_capacitance,
        const T* area,
        const I* cv_to_cell,
        const T* dt_cell,
        unsigned n)
{
    const unsigned tid = threadIdx.x + blockDim.x*blockIdx.x;

    if (tid<n) {
        auto cid = cv_to_cell[tid];
        auto dt = dt_cell[cid];

        // Note: dt==0 case is expected only at the end of a mindelay/2
        // integration period, and consequently divergence is unlikely
        // to be a peformance problem.

        if (dt>0) {
            // The 1e-3 is a constant of proportionality required to ensure that the
            // conductance (gi) values have units μS (micro-Siemens).
            // See the model documentation in docs/model for more information.
            T factor = 1e-3/dt;

            auto gi = factor * cv_capacitance[tid];
            d[tid] = gi + invariant_d[tid];
            rhs[tid] = gi*voltage[tid] - T(1e-3)*area[tid]*current[tid];
        }
        else {
            d[tid] = 0;
            rhs[tid] = voltage[tid];
        }
    }
}

/// GPU implementatin of Hines matrix assembly
/// Interleaved layout
/// For a given time step size dt
///     - use the precomputed alpha and alpha_d values to construct the diagonal
///       and off diagonal of the symmetric Hines matrix.
///     - compute the RHS of the linear system to solve
template <typename T, typename I, unsigned BlockWidth, unsigned LoadWidth, unsigned Threads>
__global__
void assemble_matrix_interleaved(
        T* d,
        T* rhs,
        const T* invariant_d,
        const T* voltage,
        const T* current,
        const T* cv_capacitance,
        const T* area,
        const I* sizes,
        const I* starts,
        const I* matrix_to_cell,
        const T* dt_cell,
        unsigned padded_size, unsigned num_mtx)
{
    static_assert(BlockWidth*LoadWidth==Threads,
        "number of threads must equal number of values to process per block");
    __shared__ T buffer_v[Threads];
    __shared__ T buffer_i[Threads];

    const unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
    const unsigned lid = threadIdx.x;

    const unsigned mtx_id   = tid/LoadWidth;
    const unsigned mtx_lane = tid - mtx_id*LoadWidth;

    const unsigned blk_id   = tid/(BlockWidth*LoadWidth);
    const unsigned blk_row  = lid/BlockWidth;
    const unsigned blk_lane = lid - blk_row*BlockWidth;

    const unsigned blk_pos  = LoadWidth*blk_lane + blk_row;

    const bool do_load  = mtx_id<num_mtx;

    unsigned load_pos  = do_load? starts[mtx_id] + mtx_lane     : 0;
    const unsigned end = do_load? starts[mtx_id] + sizes[mtx_id]: 0;
    unsigned store_pos = blk_id*BlockWidth*padded_size + (blk_row*BlockWidth + blk_lane);

    const unsigned max_size = sizes[0];

    T factor = 0;
    T dt = 0;
    const unsigned permuted_cid = blk_id*BlockWidth + blk_lane;

    if (permuted_cid<num_mtx) {
        auto cid = matrix_to_cell[permuted_cid];
        dt = dt_cell[cid];

        // The 1e-3 is a constant of proportionality required to ensure that the
        // conductance (gi) values have units μS (micro-Siemens).
        // See the model documentation in docs/model for more information.

        factor = dt>0? 1e-3/dt: 0;
    }

    for (unsigned j=0u; j<max_size; j+=LoadWidth) {
        if (do_load && load_pos<end) {
            buffer_v[lid] = voltage[load_pos];
            buffer_i[lid] = current[load_pos];
        }

        __syncthreads();

        if (j+blk_row<padded_size) {
            const auto gi = factor * cv_capacitance[store_pos];

            if (dt>0) {
                d[store_pos]   = (gi + invariant_d[store_pos]);
                rhs[store_pos] = (gi*buffer_v[blk_pos] - T(1e-3)*area[store_pos]*buffer_i[blk_pos]);
            }
            else {
                d[store_pos]   = 0;
                rhs[store_pos] = buffer_v[blk_pos];
            }
        }

        __syncthreads();

        store_pos += LoadWidth*BlockWidth;
        load_pos  += LoadWidth;
    }
}

} // namespace kernels

void assemble_matrix_flat(
        fvm_value_type* d,
        fvm_value_type* rhs,
        const fvm_value_type* invariant_d,
        const fvm_value_type* voltage,
        const fvm_value_type* current,
        const fvm_value_type* cv_capacitance,
        const fvm_value_type* area,
        const fvm_index_type* cv_to_cell,
        const fvm_value_type* dt_cell,
        unsigned n,
        hipStream_t* stream)
{
    constexpr unsigned block_dim = 128;
    const unsigned grid_dim = impl::block_count(n, block_dim);

    kernels::assemble_matrix_flat
        <fvm_value_type, fvm_index_type>
        <<<grid_dim, block_dim, 0, *stream>>>
        (d, rhs, invariant_d, voltage, current, cv_capacitance,
         area, cv_to_cell, dt_cell, n);
}

//template <typename T, typename I, unsigned BlockWidth, unsigned LoadWidth, unsigned Threads>
void assemble_matrix_interleaved(
    fvm_value_type* d,
    fvm_value_type* rhs,
    const fvm_value_type* invariant_d,
    const fvm_value_type* voltage,
    const fvm_value_type* current,
    const fvm_value_type* cv_capacitance,
    const fvm_value_type* area,
    const fvm_index_type* sizes,
    const fvm_index_type* starts,
    const fvm_index_type* matrix_to_cell,
    const fvm_value_type* dt_cell,
    unsigned padded_size, unsigned num_mtx,
    hipStream_t* stream)
{
    constexpr unsigned bd = impl::matrices_per_block();
    constexpr unsigned lw = impl::load_width();
    constexpr unsigned block_dim = bd*lw;

    // The number of threads is threads_per_matrix*num_mtx
    const unsigned grid_dim = impl::block_count(num_mtx*lw, block_dim);

    kernels::assemble_matrix_interleaved
        <fvm_value_type, fvm_index_type, bd, lw, block_dim>
        <<<grid_dim, block_dim, 0, *stream>>>
        (d, rhs, invariant_d, voltage, current, cv_capacitance, area,
         sizes, starts, matrix_to_cell,
         dt_cell, padded_size, num_mtx);
}

} // namespace gpu
} // namespace arb
