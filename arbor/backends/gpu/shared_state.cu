#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <cstdint>

#include <backends/event.hpp>
#include <backends/multi_event_stream_state.hpp>

#include "gpu_api.hpp"
#include "gpu_common.hpp"

namespace arb {
namespace gpu {

namespace kernel {

template <typename T>
__global__ void update_time_to_impl(unsigned n,
                                    T* __restrict__ const time_to,
                                    const T* __restrict__ const time,
                                    T dt,
                                    T tmax) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto t = time[i]+dt;
        time_to[i] = t<tmax? t: tmax;
    }
}

template <typename T, typename I>
__global__ void add_gj_current_impl(unsigned n,
                                    const T* __restrict__ const gj_info,
                                    const I* __restrict__ const voltage,
                                    I* __restrict__ const current_density) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto gj = gj_info[i];
        auto curr = gj.weight * (voltage[gj.loc.second] - voltage[gj.loc.first]); // nA

        gpu_atomic_sub(current_density + gj.loc.first, curr);
    }
}

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           fvm_value_type v) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        x[i] += v;
    }
}

template <typename T, typename I>
__global__ void set_dt_impl(      T* __restrict__ dt_intdom,
                            const T* __restrict__ time_to,
                            const T* __restrict__ time,
                            const unsigned ncomp,
                                  T* __restrict__ dt_comp,
                            const I* __restrict__ cv_to_intdom) {
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < ncomp) {
        const auto ind = cv_to_intdom[idx];
        const auto dt = time_to[ind] - time[ind];
        dt_intdom[ind] = dt;
        dt_comp[idx] = dt;
    }
}

__global__ void take_samples_impl(
    multi_event_stream_state<raw_probe_info> s,
    const fvm_value_type* __restrict__ const time,
    fvm_value_type* __restrict__ const sample_time,
    fvm_value_type* __restrict__ const sample_value)
{
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<s.n) {
        auto begin = s.ev_data+s.begin_offset[i];
        auto end = s.ev_data+s.end_offset[i];
        for (auto p = begin; p!=end; ++p) {
            sample_time[p->offset] = time[i];
            sample_value[p->offset] = *p->handle;
        }
    }
}

__global__ void reduce_impl(
        const fvm_value_type* __restrict__ local_i,
        const fvm_value_type* __restrict__ local_g,
        fvm_value_type* __restrict__ global_i,
        fvm_value_type* __restrict__ global_g,
        const fvm_index_type* __restrict__ reduction_partition,
        fvm_size_type ncv,
        fvm_size_type warp_size)
{
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<ncv) {
        fvm_value_type sum_i = 0, sum_g = 0;
        unsigned warp     = i/warp_size;
        unsigned warp_idx = i%warp_size; 
        auto start = reduction_partition[warp];
        auto end   = reduction_partition[warp+1];

        for (unsigned offset = start; offset < end; offset += warp_size) {
            sum_i += local_i[warp_idx + offset];
            sum_g += local_g[warp_idx + offset];
        }
        global_i[i] = sum_i;
        global_g[i] = sum_g;
    }
}

} // namespace kernel

using impl::block_count;

void add_scalar(std::size_t n, fvm_value_type* data, fvm_value_type v) {
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::add_scalar<<<nblock, block_dim>>>(n, data, v);
}

void update_time_to_impl(
    std::size_t n, fvm_value_type* time_to, const fvm_value_type* time,
    fvm_value_type dt, fvm_value_type tmax)
{
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::update_time_to_impl<<<nblock, block_dim>>>(n, time_to, time, dt, tmax);
}

void set_dt_impl(
    fvm_size_type nintdom, fvm_size_type ncomp, fvm_value_type* dt_intdom, fvm_value_type* dt_comp,
    const fvm_value_type* time_to, const fvm_value_type* time, const fvm_index_type* cv_to_intdom)
{
    if (!nintdom || !ncomp) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(ncomp, block_dim);
    kernel::set_dt_impl<<<nblock, block_dim>>>(dt_intdom, time_to, time, ncomp, dt_comp, cv_to_intdom);
}

void add_gj_current_impl(
    fvm_size_type n_gj, const fvm_gap_junction* gj_info, const fvm_value_type* voltage, fvm_value_type* current_density)
{
    if (!n_gj) return;

    constexpr int block_dim = 128;
    int nblock = block_count(n_gj, block_dim);
    kernel::add_gj_current_impl<<<nblock, block_dim>>>(n_gj, gj_info, voltage, current_density);
}

void take_samples_impl(
    const multi_event_stream_state<raw_probe_info>& s,
    const fvm_value_type* time, fvm_value_type* sample_time, fvm_value_type* sample_value)
{
    if (!s.n_streams()) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(s.n_streams(), block_dim);
    kernel::take_samples_impl<<<nblock, block_dim>>>(s, time, sample_time, sample_value);
}


void reduce_impl(
    const fvm_value_type* local_i,
    const fvm_value_type* local_g,
    fvm_value_type* global_i,
    fvm_value_type* global_g,
    const fvm_index_type* reduction_part,
    fvm_size_type ncv)
{
    if (!ncv) return;

    constexpr int block_dim = 128;
    constexpr int warp_size = impl::threads_per_warp();
    const int nblock = block_count(ncv, block_dim);
    kernel::reduce_impl<<<nblock, block_dim>>>(local_i, local_g, global_i, global_g, reduction_part, ncv, warp_size);
}

} // namespace gpu
} // namespace arb
